#include "SU.hh"

#include "scuda.h"
#include "squad.h"
#include "sphoton.h"

#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/count.h>


template<typename T>
T* SU::upload(const T* h, unsigned num_items )
{
    T* d ;
    hipMalloc(&d, num_items*sizeof(T));
    hipMemcpy(d, h, num_items*sizeof(T), hipMemcpyHostToDevice);
    return d ; 
}

template SYSRAP_API quad4*   SU::upload(const quad4* , unsigned ); 


/**
SU::deprecated_select_copy_device_to_host
--------------------------------------------

1. apply thrust::count_if to *d* with *selector* functor yielding *num_select* 
2. allocate *d_select* with num_select*sizeof(T) bytes
3. thrust::copy_if from *d* to *d_select* using the *selector* functor
4. host new T[num_select] allocation 
5. copies from *d_select* to the *num_select* host array *h* using the selector 

This API is deprecated because its awkward as the number selected is not known when making the call.
For example it would be difficult to populate an NP array using this without 
making copies. 

**/

template<typename T>
void SU::deprecated_select_copy_device_to_host( T** h, unsigned& num_select,  T* d, unsigned num_d, const qselector<T>& selector  )
{   
    thrust::device_ptr<T> td(d);
    num_select = thrust::count_if(td, td+num_d , selector );
    std::cout << " num_select " << num_select << std::endl ;
    
    T* d_select ;   
    hipMalloc(&d_select,     num_select*sizeof(T));
    thrust::device_ptr<T> td_select(d_select);
    
    thrust::copy_if(td, td+num_d , td_select, selector );
    
    *h = new T[num_select] ; 
    hipMemcpy(*h, d_select, num_select*sizeof(T), hipMemcpyDeviceToHost);
}
template SYSRAP_API void     SU::deprecated_select_copy_device_to_host( quad4** h, unsigned& ,  quad4* , unsigned , const qselector<quad4>&  ); 


/**
SU::count_if
------------------

1. apply thrust::count_if to *d* with *selector* functor yielding *num_select* 

**/

template<typename T>
unsigned SU::count_if( const T* d, unsigned num_d,  const qselector<T>& selector )
{
    thrust::device_ptr<const T> td(d);
    return thrust::count_if(td, td+num_d , selector );
}

template SYSRAP_API unsigned SU::count_if( const quad4* , unsigned, const qselector<quad4>& ); 

/**
SU::count_if_sphoton
----------------------

NB d is device side pointer 

**/

unsigned SU::count_if_sphoton( const sphoton* d, unsigned num_d,  const sphoton_selector& selector )
{
    thrust::device_ptr<const sphoton> td(d);
    return thrust::count_if(td, td+num_d , selector );
}






/**
SU::device_alloc
-------------------

1. allocates *d* with num*sizeof(T) bytes

**/

template<typename T>
T* SU::device_alloc( unsigned num  )
{
    T* d ;
    hipMalloc(&d,  num*sizeof(T));
    return d ; 
}
template SYSRAP_API char*  SU::device_alloc( unsigned ); 
template SYSRAP_API float* SU::device_alloc( unsigned ); 
template SYSRAP_API quad4* SU::device_alloc( unsigned ); 





template<typename T>
void SU::device_zero( T* d, unsigned num )
{
    hipMemset(d, 0,  num*sizeof(T));
}
template SYSRAP_API void SU::device_zero( quad4*, unsigned ); 


/**
SU::copy_if_device_to_device_presized
-----------------------------------------

The d_select array must be presized to fit the selection, determine the size using *select_count* with the same selector. 

2. thrust::copy_if from *d* to *d_select* using the selector functor
3. copies from *d_select* to the *num_select* presized host array *h* using the selector

**/

template<typename T>
void SU::copy_if_device_to_device_presized( T* d_select, const T* d, unsigned num_d, const qselector<T>& selector )
{
    thrust::device_ptr<const T> td(d);
    thrust::device_ptr<T> td_select(d_select);
    thrust::copy_if(td, td+num_d , td_select, selector );
}

template SYSRAP_API void SU::copy_if_device_to_device_presized( quad4*, const quad4*, unsigned, const qselector<quad4>& ); 


void SU::copy_if_device_to_device_presized_sphoton( sphoton* d_select, const sphoton* d, unsigned num_d, const sphoton_selector& selector )
{
    thrust::device_ptr<const sphoton> td(d);
    thrust::device_ptr<sphoton> td_select(d_select);
    thrust::copy_if(td, td+num_d , td_select, selector );
}



template<typename T>
void SU::copy_device_to_host_presized( T* h, const T* d, unsigned num  )
{
    hipMemcpy(h, d, num*sizeof(T), hipMemcpyDeviceToHost);
}
template SYSRAP_API void SU::copy_device_to_host_presized( quad4*, const quad4*, unsigned ); 





// TODO: error check all these cuda operations following QU


char* SU::device_alloc_sizeof( unsigned num, unsigned sizeof_item )
{
    char* d ; 
    hipMalloc(&d,  num*sizeof_item );
    return d ; 
}

void SU::copy_host_to_device_sizeof( char* d, const char* h, unsigned num, unsigned sizeof_item )
{
    hipMemcpy(d, h, num*sizeof_item, hipMemcpyHostToDevice);
}

void SU::copy_device_to_host_sizeof( char* h, const char* d, unsigned num, unsigned sizeof_item )
{
    hipMemcpy(h, d, num*sizeof_item, hipMemcpyDeviceToHost);
}

char* SU::upload_array_sizeof(const char* h, unsigned num_items, unsigned sizeof_item ) // static
{
    char* d = nullptr ; 
    hipMalloc(reinterpret_cast<void**>( &d ), num_items*sizeof_item ); 
    hipMemcpy(reinterpret_cast<void*>( d ), h, sizeof_item*num_items, hipMemcpyHostToDevice ); 
    return d ; 
}


